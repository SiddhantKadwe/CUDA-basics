//-----------------------------------------------
//       Created by Siddhant Kadwe
//       -Generates an array of size 64
//        and squares the elements and
//        saves it in another array
//-----------------------------------------------


#include <hip/hip_runtime.h>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void square(float *d_out, float *d_in) {
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f;
}

int main(int argc, char ** argv) {
    const int ARRAY_SIZE = 64;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // generate the input array on the host
    float h_in[ARRAY_SIZE];
    for (int i = 0; i < ARRAY_SIZE; i++) {
        h_in[i] = float(i);
    }
    float h_out[ARRAY_SIZE];

    // declare GPU memory pointers
    float * d_in;
    float * d_out;

    // allocate GPU memory
    gpuErrchk(hipMalloc((void **) &d_in, ARRAY_BYTES));
    gpuErrchk(hipMalloc((void **) &d_out, ARRAY_BYTES));

    // transfer the array to the GPU
    gpuErrchk(hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice));

    // launch the kernel
    (square<<<1, ARRAY_SIZE>>>(d_out, d_in));

    // copy back the result array to the CPU
    gpuErrchk(hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost));

    // print out the resulting array
    for(int i = 0; i < ARRAY_SIZE; i++) {
        printf("%f", h_out[i]);
        printf(((i%4)!=3) ? "\t" : "\n");
    }

    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}